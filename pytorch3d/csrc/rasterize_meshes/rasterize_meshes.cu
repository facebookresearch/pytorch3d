#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <thrust/tuple.h>
#include <cstdio>
#include <tuple>
#include "rasterize_points/rasterization_utils.cuh"
#include "utils/float_math.cuh"
#include "utils/geometry_utils.cuh"

namespace {
// A structure for holding details about a pixel.
struct Pixel {
  float z;
  int64_t idx; // idx of face
  float dist; // abs distance of pixel to face
  float3 bary;
};

__device__ bool operator<(const Pixel& a, const Pixel& b) {
  return a.z < b.z || (a.z == b.z && a.idx < b.idx);
}

// Get the xyz coordinates of the three vertices for the face given by the
// index face_idx into face_verts.
__device__ thrust::tuple<float3, float3, float3> GetSingleFaceVerts(
    const float* face_verts,
    int face_idx) {
  const float x0 = face_verts[face_idx * 9 + 0];
  const float y0 = face_verts[face_idx * 9 + 1];
  const float z0 = face_verts[face_idx * 9 + 2];
  const float x1 = face_verts[face_idx * 9 + 3];
  const float y1 = face_verts[face_idx * 9 + 4];
  const float z1 = face_verts[face_idx * 9 + 5];
  const float x2 = face_verts[face_idx * 9 + 6];
  const float y2 = face_verts[face_idx * 9 + 7];
  const float z2 = face_verts[face_idx * 9 + 8];

  const float3 v0xyz = make_float3(x0, y0, z0);
  const float3 v1xyz = make_float3(x1, y1, z1);
  const float3 v2xyz = make_float3(x2, y2, z2);

  return thrust::make_tuple(v0xyz, v1xyz, v2xyz);
}

// Get the min/max x/y/z values for the face given by vertices v0, v1, v2.
__device__ thrust::tuple<float2, float2, float2>
GetFaceBoundingBox(float3 v0, float3 v1, float3 v2) {
  const float xmin = FloatMin3(v0.x, v1.x, v2.x);
  const float ymin = FloatMin3(v0.y, v1.y, v2.y);
  const float zmin = FloatMin3(v0.z, v1.z, v2.z);
  const float xmax = FloatMax3(v0.x, v1.x, v2.x);
  const float ymax = FloatMax3(v0.y, v1.y, v2.y);
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);

  return thrust::make_tuple(
      make_float2(xmin, xmax),
      make_float2(ymin, ymax),
      make_float2(zmin, zmax));
}

// Check if the point (px, py) lies outside the face bounding box face_bbox.
// Return true if the point is outside.
__device__ bool CheckPointOutsideBoundingBox(
    float3 v0,
    float3 v1,
    float3 v2,
    float blur_radius,
    float2 pxy) {
  const auto bbox = GetFaceBoundingBox(v0, v1, v2);
  const float2 xlims = thrust::get<0>(bbox);
  const float2 ylims = thrust::get<1>(bbox);
  const float2 zlims = thrust::get<2>(bbox);

  const float x_min = xlims.x - blur_radius;
  const float y_min = ylims.x - blur_radius;
  const float x_max = xlims.y + blur_radius;
  const float y_max = ylims.y + blur_radius;

  // Faces with at least one vertex behind the camera won't render correctly
  // and should be removed or clipped before calling the rasterizer
  const bool z_invalid = zlims.x < kEpsilon;

  // Check if the current point is oustside the triangle bounding box.
  return (
      pxy.x > x_max || pxy.x < x_min || pxy.y > y_max || pxy.y < y_min ||
      z_invalid);
}

// This function checks if a pixel given by xy location pxy lies within the
// face with index face_idx in face_verts. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the faces which intersect
// with this pixel sorted by closest z distance. If the point pxy lies in the
// face, the list (q) is updated and re-orderered in place. In addition
// the auxiliary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizeMeshesNaiveCudaKernel and
// RasterizeMeshesFineCudaKernel.
template <typename FaceQ>
__device__ void CheckPixelInsideFace(
    const float* face_verts, // (F, 3, 3)
    const int64_t* clipped_faces_neighbor_idx, // (F,)
    const int face_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    FaceQ& q,
    const float blur_radius,
    const float2 pxy, // Coordinates of the pixel
    const int K,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  const auto v012 = GetSingleFaceVerts(face_verts, face_idx);
  const float3 v0 = thrust::get<0>(v012);
  const float3 v1 = thrust::get<1>(v012);
  const float3 v2 = thrust::get<2>(v012);

  // Only need xy for barycentric coordinates and distance calculations.
  const float2 v0xy = make_float2(v0.x, v0.y);
  const float2 v1xy = make_float2(v1.x, v1.y);
  const float2 v2xy = make_float2(v2.x, v2.y);

  // Perform checks and skip if:
  // 1. the face is behind the camera
  // 2. the face is facing away from the camera
  // 3. the face has very small face area
  // 4. the pixel is outside the face bbox
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);
  const bool outside_bbox = CheckPointOutsideBoundingBox(
      v0, v1, v2, sqrt(blur_radius), pxy); // use sqrt of blur for bbox
  const float face_area = EdgeFunctionForward(v0xy, v1xy, v2xy);
  // Check if the face is visible to the camera.
  const bool back_face = face_area < 0.0;
  const bool zero_face_area =
      (face_area <= kEpsilon && face_area >= -1.0f * kEpsilon);

  if (zmax < 0 || (cull_backfaces && back_face) || outside_bbox ||
      zero_face_area) {
    return;
  }

  // Calculate barycentric coords and euclidean dist to triangle.
  const float3 p_bary0 = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
  const float3 p_bary = !perspective_correct
      ? p_bary0
      : BarycentricPerspectiveCorrectionForward(p_bary0, v0.z, v1.z, v2.z);
  const float3 p_bary_clip =
      !clip_barycentric_coords ? p_bary : BarycentricClipForward(p_bary);

  const float pz =
      p_bary_clip.x * v0.z + p_bary_clip.y * v1.z + p_bary_clip.z * v2.z;

  if (pz < 0) {
    return; // Face is behind the image plane.
  }

  // Get abs squared distance
  const float dist = PointTriangleDistanceForward(pxy, v0xy, v1xy, v2xy);

  // Use the unclipped bary coordinates to determine if the point is inside the
  // face.
  const bool inside = p_bary.x > 0.0f && p_bary.y > 0.0f && p_bary.z > 0.0f;
  const float signed_dist = inside ? -dist : dist;
  // Check if pixel is outside blur region
  if (!inside && dist >= blur_radius) {
    return;
  }

  // Handle the case where a face (f) partially behind the image plane is
  // clipped to a quadrilateral and then split into two faces (t1, t2). In this
  // case we:
  // 1. Find the index of the neighboring face (e.g. for t1 need index of t2)
  // 2. Check if the neighboring face (t2) is already in the top K faces
  // 3. If yes, compare the distance of the pixel to t1 with the distance to t2.
  // 4. If dist_t1 < dist_t2, overwrite the values for t2 in the top K faces.
  const int neighbor_idx = clipped_faces_neighbor_idx[face_idx];
  int neighbor_idx_top_k = -1;

  // Check if neighboring face is already in the top K.
  // -1 is the fill value in clipped_faces_neighbor_idx
  if (neighbor_idx != -1) {
    // Only need to loop until q_size.
    for (int i = 0; i < q_size; i++) {
      if (q[i].idx == neighbor_idx) {
        neighbor_idx_top_k = i;
        break;
      }
    }
  }
  // If neighbor idx is not -1 then it is in the top K struct.
  if (neighbor_idx_top_k != -1) {
    // If dist of current face is less than neighbor then overwrite the
    // neighbor face values in the top K struct.
    float neighbor_dist = abs(q[neighbor_idx_top_k].dist);
    if (dist < neighbor_dist) {
      // Overwrite the neighbor face values
      q[neighbor_idx_top_k] = {pz, face_idx, signed_dist, p_bary_clip};

      // If pz > q_max then overwrite the max values and index of the max.
      // q_size stays the same.
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = neighbor_idx_top_k;
      }
    }
  } else {
    // Handle as a normal face
    if (q_size < K) {
      // Just insert it.
      q[q_size] = {pz, face_idx, signed_dist, p_bary_clip};
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = q_size;
      }
      q_size++;
    } else if (pz < q_max_z) {
      // Overwrite the old max, and find the new max.
      q[q_max_idx] = {pz, face_idx, signed_dist, p_bary_clip};
      q_max_z = pz;
      for (int i = 0; i < K; i++) {
        if (q[i].z > q_max_z) {
          q_max_z = q[i].z;
          q_max_idx = i;
        }
      }
    }
  }
}

} // namespace

// ****************************************************************************
// *                          NAIVE RASTERIZATION                      *
// ****************************************************************************
__global__ void RasterizeMeshesNaiveCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const int64_t* clipped_faces_neighbor_idx,
    const float blur_radius,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces,
    const int N,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs,
    float* zbuf,
    float* pix_dists,
    float* bary) {
  // Simple version: One thread per output pixel
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = tid; i < N * H * W; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (H * W); // batch index.
    const int pix_idx = i % (H * W);

    // Reverse ordering of X and Y axes
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    // screen coordinates to ndc coordinates of pixel.
    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);
    const float2 pxy = make_float2(xf, yf);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the faces.
    const int64_t face_start_idx = mesh_to_face_first_idx[n];
    const int64_t face_stop_idx = face_start_idx + num_faces_per_mesh[n];

    // Loop through the faces in the mesh.
    for (int f = face_start_idx; f < face_stop_idx; ++f) {
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.

      CheckPixelInsideFace(
          face_verts,
          clipped_faces_neighbor_idx,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct,
          clip_barycentric_coords,
          cull_backfaces);
    }

    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);
    int idx = n * H * W * K + pix_idx * K;

    for (int k = 0; k < q_size; ++k) {
      face_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist;
      bary[(idx + k) * 3 + 0] = q[k].bary.x;
      bary[(idx + k) * 3 + 1] = q[k].bary.y;
      bary[(idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
RasterizeMeshesNaiveCuda(
    const at::Tensor& face_verts,
    const at::Tensor& mesh_to_faces_packed_first_idx,
    const at::Tensor& num_faces_per_mesh,
    const at::Tensor& clipped_faces_neighbor_idx,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int num_closest,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");

  TORCH_CHECK(
      num_faces_per_mesh.size(0) == mesh_to_faces_packed_first_idx.size(0),
      "num_faces_per_mesh must have save size first dimension as mesh_to_faces_packed_first_idx");

  TORCH_CHECK(
      clipped_faces_neighbor_idx.size(0) == face_verts.size(0),
      "clipped_faces_neighbor_idx must have save size first dimension as face_verts");

  if (num_closest > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      mesh_to_faces_packed_first_idx_t{
          mesh_to_faces_packed_first_idx, "mesh_to_faces_packed_first_idx", 2},
      num_faces_per_mesh_t{num_faces_per_mesh, "num_faces_per_mesh", 3},
      clipped_faces_neighbor_idx_t{
          clipped_faces_neighbor_idx, "clipped_faces_neighbor_idx", 4};
  at::CheckedFrom c = "RasterizeMeshesNaiveCuda";
  at::checkAllSameGPU(
      c,
      {face_verts_t,
       mesh_to_faces_packed_first_idx_t,
       num_faces_per_mesh_t,
       clipped_faces_neighbor_idx_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = num_faces_per_mesh.size(0); // batch size.
  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);
  const int K = num_closest;

  auto long_opts = num_faces_per_mesh.options().dtype(at::kLong);
  auto float_opts = face_verts.options().dtype(at::kFloat);

  at::Tensor face_idxs = at::full({N, H, W, K}, -1, long_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor bary = at::full({N, H, W, K, 3}, -1, float_opts);

  if (face_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesNaiveCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_faces_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      clipped_faces_neighbor_idx.contiguous().data_ptr<int64_t>(),
      blur_radius,
      perspective_correct,
      clip_barycentric_coords,
      cull_backfaces,
      N,
      H,
      W,
      K,
      face_idxs.data_ptr<int64_t>(),
      zbuf.data_ptr<float>(),
      pix_dists.data_ptr<float>(),
      bary.data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO: benchmark parallelizing over faces_verts instead of over pixels.
__global__ void RasterizeMeshesBackwardCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int64_t* pix_to_face, // (N, H, W, K)
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const int N,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_bary, // (N, H, W, K, 3)
    const float* grad_dists, // (N, H, W, K)
    float* grad_face_verts) { // (F, 3, 3)

  // Parallelize over each pixel in images of
  // size H * W, for each image in the batch of size N.
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int t_i = tid; t_i < N * H * W; t_i += num_threads) {
    // Convert linear index to 3D index
    const int n = t_i / (H * W); // batch index.
    const int pix_idx = t_i % (H * W);

    // Reverse ordering of X and Y axes.
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);
    const float2 pxy = make_float2(xf, yf);

    // Loop over all the faces for this pixel.
    for (int k = 0; k < K; k++) {
      // Index into (N, H, W, K, :) grad tensors
      // pixel index + top k index
      int i = n * H * W * K + pix_idx * K + k;

      const int f = pix_to_face[i];
      if (f < 0) {
        continue; // padded face.
      }
      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Only neex xy for barycentric coordinate and distance calculations.
      const float2 v0xy = make_float2(v0.x, v0.y);
      const float2 v1xy = make_float2(v1.x, v1.y);
      const float2 v2xy = make_float2(v2.x, v2.y);

      // Get upstream gradients for the face.
      const float grad_dist_upstream = grad_dists[i];
      const float grad_zbuf_upstream = grad_zbuf[i];
      const float grad_bary_upstream_w0 = grad_bary[i * 3 + 0];
      const float grad_bary_upstream_w1 = grad_bary[i * 3 + 1];
      const float grad_bary_upstream_w2 = grad_bary[i * 3 + 2];
      const float3 grad_bary_upstream = make_float3(
          grad_bary_upstream_w0, grad_bary_upstream_w1, grad_bary_upstream_w2);

      const float3 b_w = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
      const float3 b_pp = !perspective_correct
          ? b_w
          : BarycentricPerspectiveCorrectionForward(b_w, v0.z, v1.z, v2.z);

      const float3 b_w_clip =
          !clip_barycentric_coords ? b_pp : BarycentricClipForward(b_pp);

      const bool inside = b_pp.x > 0.0f && b_pp.y > 0.0f && b_pp.z > 0.0f;
      const float sign = inside ? -1.0f : 1.0f;

      auto grad_dist_f = PointTriangleDistanceBackward(
          pxy, v0xy, v1xy, v2xy, sign * grad_dist_upstream);
      const float2 ddist_d_v0 = thrust::get<1>(grad_dist_f);
      const float2 ddist_d_v1 = thrust::get<2>(grad_dist_f);
      const float2 ddist_d_v2 = thrust::get<3>(grad_dist_f);

      // Upstream gradient for barycentric coords from zbuf calculation:
      // zbuf = bary_w0 * z0 + bary_w1 * z1 + bary_w2 * z2
      // Therefore
      // d_zbuf/d_bary_w0 = z0
      // d_zbuf/d_bary_w1 = z1
      // d_zbuf/d_bary_w2 = z2
      const float3 d_zbuf_d_bwclip = make_float3(v0.z, v1.z, v2.z);

      // Total upstream barycentric gradients are the sum of
      // external upstream gradients and contribution from zbuf.
      const float3 grad_bary_f_sum =
          (grad_bary_upstream + grad_zbuf_upstream * d_zbuf_d_bwclip);

      float3 grad_bary0 = grad_bary_f_sum;

      if (clip_barycentric_coords) {
        grad_bary0 = BarycentricClipBackward(b_w, grad_bary_f_sum);
      }

      float dz0_persp = 0.0f, dz1_persp = 0.0f, dz2_persp = 0.0f;
      if (perspective_correct) {
        auto perspective_grads = BarycentricPerspectiveCorrectionBackward(
            b_w, v0.z, v1.z, v2.z, grad_bary0);
        grad_bary0 = thrust::get<0>(perspective_grads);
        dz0_persp = thrust::get<1>(perspective_grads);
        dz1_persp = thrust::get<2>(perspective_grads);
        dz2_persp = thrust::get<3>(perspective_grads);
      }

      auto grad_bary_f =
          BarycentricCoordsBackward(pxy, v0xy, v1xy, v2xy, grad_bary0);
      const float2 dbary_d_v0 = thrust::get<1>(grad_bary_f);
      const float2 dbary_d_v1 = thrust::get<2>(grad_bary_f);
      const float2 dbary_d_v2 = thrust::get<3>(grad_bary_f);

      atomicAdd(grad_face_verts + f * 9 + 0, dbary_d_v0.x + ddist_d_v0.x);
      atomicAdd(grad_face_verts + f * 9 + 1, dbary_d_v0.y + ddist_d_v0.y);
      atomicAdd(
          grad_face_verts + f * 9 + 2,
          grad_zbuf_upstream * b_w_clip.x + dz0_persp);
      atomicAdd(grad_face_verts + f * 9 + 3, dbary_d_v1.x + ddist_d_v1.x);
      atomicAdd(grad_face_verts + f * 9 + 4, dbary_d_v1.y + ddist_d_v1.y);
      atomicAdd(
          grad_face_verts + f * 9 + 5,
          grad_zbuf_upstream * b_w_clip.y + dz1_persp);
      atomicAdd(grad_face_verts + f * 9 + 6, dbary_d_v2.x + ddist_d_v2.x);
      atomicAdd(grad_face_verts + f * 9 + 7, dbary_d_v2.y + ddist_d_v2.y);
      atomicAdd(
          grad_face_verts + f * 9 + 8,
          grad_zbuf_upstream * b_w_clip.z + dz2_persp);
    }
  }
}

at::Tensor RasterizeMeshesBackwardCuda(
    const at::Tensor& face_verts, // (F, 3, 3)
    const at::Tensor& pix_to_face, // (N, H, W, K)
    const at::Tensor& grad_zbuf, // (N, H, W, K)
    const at::Tensor& grad_bary, // (N, H, W, K, 3)
    const at::Tensor& grad_dists, // (N, H, W, K)
    const bool perspective_correct,
    const bool clip_barycentric_coords) {
  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      pix_to_face_t{pix_to_face, "pix_to_face", 2},
      grad_zbuf_t{grad_zbuf, "grad_zbuf", 3},
      grad_bary_t{grad_bary, "grad_bary", 4},
      grad_dists_t{grad_dists, "grad_dists", 5};
  at::CheckedFrom c = "RasterizeMeshesBackwardCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, pix_to_face_t, grad_zbuf_t, grad_bary_t, grad_dists_t});
  at::checkAllSameType(
      c, {face_verts_t, grad_zbuf_t, grad_bary_t, grad_dists_t});

  // This is nondeterministic because atomicAdd
  at::globalContext().alertNotDeterministic("RasterizeMeshesBackwardCuda");

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int F = face_verts.size(0);
  const int N = pix_to_face.size(0);
  const int H = pix_to_face.size(1);
  const int W = pix_to_face.size(2);
  const int K = pix_to_face.size(3);

  at::Tensor grad_face_verts = at::zeros({F, 3, 3}, face_verts.options());

  if (grad_face_verts.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_face_verts;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesBackwardCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      pix_to_face.contiguous().data_ptr<int64_t>(),
      perspective_correct,
      clip_barycentric_coords,
      N,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_bary.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_face_verts.data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return grad_face_verts;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************
__global__ void RasterizeMeshesFineCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int32_t* bin_faces, // (N, BH, BW, T)
    const int64_t* clipped_faces_neighbor_idx, // (F,)
    const float blur_radius,
    const int bin_size,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces,
    const int N,
    const int BH,
    const int BW,
    const int M,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists, // (N, H, W, K)
    float* bary // (N, H, W, K, 3)
) {
  // This can be more than H * W if H or W are not divisible by bin_size.
  int num_pixels = N * BH * BW * bin_size * bin_size;
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from faces and bin_faces.
    int i = pid;
    const int n = i / (BH * BW * bin_size * bin_size);
    i %= BH * BW * bin_size * bin_size;
    // bin index y
    const int by = i / (BW * bin_size * bin_size);
    i %= BW * bin_size * bin_size;
    // bin index y
    const int bx = i / (bin_size * bin_size);
    // pixel within the bin
    i %= bin_size * bin_size;

    // Pixel x, y indices
    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= H || xi >= W)
      continue;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    const float2 pxy = make_float2(xf, yf);

    // This part looks like the naive rasterization kernel, except we use
    // bin_faces to only look at a subset of faces already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    for (int m = 0; m < M; m++) {
      const int f = bin_faces[n * BH * BW * M + by * BW * M + bx * M + m];
      if (f < 0) {
        continue; // bin_faces uses -1 as a sentinal value.
      }
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.
      CheckPixelInsideFace(
          face_verts,
          clipped_faces_neighbor_idx,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct,
          clip_barycentric_coords,
          cull_backfaces);
    }

    // Now we've looked at all the faces for this bin, so we can write
    // output for the current pixel.
    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis so that
    // in the image +Y is pointing up and +X is pointing left.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const int pix_idx = n * H * W * K + yidx * W * K + xidx * K;
    for (int k = 0; k < q_size; k++) {
      face_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist;
      bary[(pix_idx + k) * 3 + 0] = q[k].bary.x;
      bary[(pix_idx + k) * 3 + 1] = q[k].bary.y;
      bary[(pix_idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
RasterizeMeshesFineCuda(
    const at::Tensor& face_verts,
    const at::Tensor& bin_faces,
    const at::Tensor& clipped_faces_neighbor_idx,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int bin_size,
    const int faces_per_pixel,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");
  TORCH_CHECK(bin_faces.ndimension() == 4, "bin_faces must have 4 dimensions");
  TORCH_CHECK(
      clipped_faces_neighbor_idx.size(0) == face_verts.size(0),
      "clipped_faces_neighbor_idx must have the same first dimension as face_verts");

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      bin_faces_t{bin_faces, "bin_faces", 2},
      clipped_faces_neighbor_idx_t{
          clipped_faces_neighbor_idx, "clipped_faces_neighbor_idx", 3};
  at::CheckedFrom c = "RasterizeMeshesFineCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, bin_faces_t, clipped_faces_neighbor_idx_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // bin_faces shape (N, BH, BW, M)
  const int N = bin_faces.size(0);
  const int BH = bin_faces.size(1);
  const int BW = bin_faces.size(2);
  const int M = bin_faces.size(3);
  const int K = faces_per_pixel;

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 150");
  }
  auto long_opts = bin_faces.options().dtype(at::kLong);
  auto float_opts = face_verts.options().dtype(at::kFloat);

  at::Tensor face_idxs = at::full({N, H, W, K}, -1, long_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor bary = at::full({N, H, W, K, 3}, -1, float_opts);

  if (face_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesFineCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      bin_faces.contiguous().data_ptr<int32_t>(),
      clipped_faces_neighbor_idx.contiguous().data_ptr<int64_t>(),
      blur_radius,
      bin_size,
      perspective_correct,
      clip_barycentric_coords,
      cull_backfaces,
      N,
      BH,
      BW,
      M,
      H,
      W,
      K,
      face_idxs.data_ptr<int64_t>(),
      zbuf.data_ptr<float>(),
      pix_dists.data_ptr<float>(),
      bary.data_ptr<float>());

  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}
