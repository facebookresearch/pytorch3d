#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// A chunk of work is blocksize-many points of P1.
// The number of potential chunks to do is N*(1+(P1-1)/blocksize)
// call (1+(P1-1)/blocksize) chunks_per_cloud
// These chunks are divided among the gridSize-many blocks.
// In block b, we work on chunks b, b+gridSize, b+2*gridSize etc .
// In chunk i, we work on cloud i/chunks_per_cloud on points starting from
// blocksize*(i%chunks_per_cloud).

template <typename scalar_t>
__global__ void BallQueryKernel(
    const at::PackedTensorAccessor64<scalar_t, 3, at::RestrictPtrTraits> p1,
    const at::PackedTensorAccessor64<scalar_t, 3, at::RestrictPtrTraits> p2,
    const at::PackedTensorAccessor64<int64_t, 1, at::RestrictPtrTraits>
        lengths1,
    const at::PackedTensorAccessor64<int64_t, 1, at::RestrictPtrTraits>
        lengths2,
    at::PackedTensorAccessor64<int64_t, 3, at::RestrictPtrTraits> idxs,
    at::PackedTensorAccessor64<scalar_t, 3, at::RestrictPtrTraits> dists,
    const int64_t K,
    const scalar_t radius2) {
  const int64_t N = p1.size(0);
  const int64_t chunks_per_cloud = (1 + (p1.size(1) - 1) / blockDim.x);
  const int64_t chunks_to_do = N * chunks_per_cloud;
  const int D = p1.size(2);

  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud; // batch_index
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t i = start_point + threadIdx.x;

    // Check if point is valid in heterogeneous tensor
    if (i >= lengths1[n]) {
      continue;
    }

    // Iterate over points in p2 until desired count is reached or
    // all points have been considered
    for (int64_t j = 0, count = 0; j < lengths2[n] && count < K; ++j) {
      // Calculate the distance between the points
      scalar_t dist2 = 0.0;
      for (int d = 0; d < D; ++d) {
        scalar_t diff = p1[n][i][d] - p2[n][j][d];
        dist2 += (diff * diff);
      }

      if (dist2 < radius2) {
        // If the point is within the radius
        // Set the value of the index to the point index
        idxs[n][i][count] = j;
        dists[n][i][count] = dist2;

        // increment the number of selected samples for the point i
        ++count;
      }
    }
  }
}

std::tuple<at::Tensor, at::Tensor> BallQueryCuda(
    const at::Tensor& p1, // (N, P1, 3)
    const at::Tensor& p2, // (N, P2, 3)
    const at::Tensor& lengths1, // (N,)
    const at::Tensor& lengths2, // (N,)
    int K,
    float radius) {
  // Check inputs are on the same device
  at::TensorArg p1_t{p1, "p1", 1}, p2_t{p2, "p2", 2},
      lengths1_t{lengths1, "lengths1", 3}, lengths2_t{lengths2, "lengths2", 4};
  at::CheckedFrom c = "BallQueryCuda";
  at::checkAllSameGPU(c, {p1_t, p2_t, lengths1_t, lengths2_t});
  at::checkAllSameType(c, {p1_t, p2_t});

  // Set the device for the kernel launch based on the device of p1
  at::cuda::CUDAGuard device_guard(p1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      p2.size(2) == p1.size(2), "Point sets must have the same last dimension");

  const int N = p1.size(0);
  const int P1 = p1.size(1);
  const int64_t K_64 = K;
  const auto radius2 = radius * radius;

  // Output tensor with indices of neighbors for each point in p1
  auto long_dtype = lengths1.options().dtype(at::kLong);
  auto idxs = at::full({N, P1, K}, -1, long_dtype);
  auto dists = at::zeros({N, P1, K}, p1.options());

  if (idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(idxs, dists);
  }

  const size_t blocks = 256;
  const size_t threads = 256;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      p1.scalar_type(), "ball_query_kernel_cuda", ([&] {
        BallQueryKernel<scalar_t><<<blocks, threads, 0, stream>>>(
            p1.packed_accessor64<scalar_t, 3, at::RestrictPtrTraits>(),
            p2.packed_accessor64<scalar_t, 3, at::RestrictPtrTraits>(),
            lengths1.packed_accessor64<int64_t, 1, at::RestrictPtrTraits>(),
            lengths2.packed_accessor64<int64_t, 1, at::RestrictPtrTraits>(),
            idxs.packed_accessor64<int64_t, 3, at::RestrictPtrTraits>(),
            dists.packed_accessor64<scalar_t, 3, at::RestrictPtrTraits>(),
            K_64,
            radius2);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  return std::make_tuple(idxs, dists);
}
