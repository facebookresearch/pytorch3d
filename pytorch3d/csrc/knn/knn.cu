#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <iostream>
#include <tuple>

#include "utils/dispatch.cuh"
#include "utils/mink.cuh"

// A chunk of work is blocksize-many points of P1.
// The number of potential chunks to do is N*(1+(P1-1)/blocksize)
// call (1+(P1-1)/blocksize) chunks_per_cloud
// These chunks are divided among the gridSize-many blocks.
// In block b, we work on chunks b, b+gridSize, b+2*gridSize etc .
// In chunk i, we work on cloud i/chunks_per_cloud on points starting from
// blocksize*(i%chunks_per_cloud).

template <typename scalar_t>
__global__ void KNearestNeighborKernelV0(
    const scalar_t* __restrict__ points1,
    const scalar_t* __restrict__ points2,
    const int64_t* __restrict__ lengths1,
    const int64_t* __restrict__ lengths2,
    scalar_t* __restrict__ dists,
    int64_t* __restrict__ idxs,
    const size_t N,
    const size_t P1,
    const size_t P2,
    const size_t D,
    const size_t K,
    const size_t norm) {
  // Store both dists and indices for knn in global memory.
  const int64_t chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int64_t chunks_to_do = N * chunks_per_cloud;
  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud;
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    int offset = n * P1 * K + p1 * K;
    int64_t length2 = lengths2[n];
    MinK<scalar_t, int64_t> mink(dists + offset, idxs + offset, K);
    for (int p2 = 0; p2 < length2; ++p2) {
      // Find the distance between points1[n, p1] and points[n, p2]
      scalar_t dist = 0;
      for (int d = 0; d < D; ++d) {
        scalar_t coord1 = points1[n * P1 * D + p1 * D + d];
        scalar_t coord2 = points2[n * P2 * D + p2 * D + d];
        scalar_t diff = coord1 - coord2;
        scalar_t norm_diff = (norm == 2) ? (diff * diff) : ((diff > 0) ? diff : -diff);
        dist += norm_diff;
      }
      mink.add(dist, p2);
    }
  }
}

template <typename scalar_t, int64_t D>
__global__ void KNearestNeighborKernelV1(
    const scalar_t* __restrict__ points1,
    const scalar_t* __restrict__ points2,
    const int64_t* __restrict__ lengths1,
    const int64_t* __restrict__ lengths2,
    scalar_t* __restrict__ dists,
    int64_t* __restrict__ idxs,
    const size_t N,
    const size_t P1,
    const size_t P2,
    const size_t K,
    const size_t norm) {
  // Same idea as the previous version, but hoist D into a template argument
  // so we can cache the current point in a thread-local array. We still store
  // the current best K dists and indices in global memory, so this should work
  // for very large K and fairly large D.
  scalar_t cur_point[D];
  const int64_t chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int64_t chunks_to_do = N * chunks_per_cloud;
  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud;
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    for (int d = 0; d < D; ++d) {
      cur_point[d] = points1[n * P1 * D + p1 * D + d];
    }
    int offset = n * P1 * K + p1 * K;
    int64_t length2 = lengths2[n];
    MinK<scalar_t, int64_t> mink(dists + offset, idxs + offset, K);
    for (int p2 = 0; p2 < length2; ++p2) {
      // Find the distance between cur_point and points[n, p2]
      scalar_t dist = 0;
      for (int d = 0; d < D; ++d) {
        scalar_t diff = cur_point[d] - points2[n * P2 * D + p2 * D + d];
        scalar_t norm_diff = (norm == 2) ? (diff * diff) : ((diff > 0) ? diff : -diff);
        dist += norm_diff;
      }
      mink.add(dist, p2);
    }
  }
}

// This is a shim functor to allow us to dispatch using DispatchKernel1D
template <typename scalar_t, int64_t D>
struct KNearestNeighborV1Functor {
  static void run(
      size_t blocks,
      size_t threads,
      const scalar_t* __restrict__ points1,
      const scalar_t* __restrict__ points2,
      const int64_t* __restrict__ lengths1,
      const int64_t* __restrict__ lengths2,
      scalar_t* __restrict__ dists,
      int64_t* __restrict__ idxs,
      const size_t N,
      const size_t P1,
      const size_t P2,
      const size_t K,
      const size_t norm) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    KNearestNeighborKernelV1<scalar_t, D><<<blocks, threads, 0, stream>>>(
        points1, points2, lengths1, lengths2, dists, idxs, N, P1, P2, K, norm);
  }
};

template <typename scalar_t, int64_t D, int64_t K>
__global__ void KNearestNeighborKernelV2(
    const scalar_t* __restrict__ points1,
    const scalar_t* __restrict__ points2,
    const int64_t* __restrict__ lengths1,
    const int64_t* __restrict__ lengths2,
    scalar_t* __restrict__ dists,
    int64_t* __restrict__ idxs,
    const int64_t N,
    const int64_t P1,
    const int64_t P2,
    const size_t norm) {
  // Same general implementation as V2, but also hoist K into a template arg.
  scalar_t cur_point[D];
  scalar_t min_dists[K];
  int min_idxs[K];
  const int64_t chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int64_t chunks_to_do = N * chunks_per_cloud;
  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud;
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    for (int d = 0; d < D; ++d) {
      cur_point[d] = points1[n * P1 * D + p1 * D + d];
    }
    int64_t length2 = lengths2[n];
    MinK<scalar_t, int> mink(min_dists, min_idxs, K);
    for (int p2 = 0; p2 < length2; ++p2) {
      scalar_t dist = 0;
      for (int d = 0; d < D; ++d) {
        int offset = n * P2 * D + p2 * D + d;
        scalar_t diff = cur_point[d] - points2[offset];
        scalar_t norm_diff = (norm == 2) ? (diff * diff) : ((diff > 0) ? diff : -diff);
        dist += norm_diff;
      }
      mink.add(dist, p2);
    }
    for (int k = 0; k < mink.size(); ++k) {
      idxs[n * P1 * K + p1 * K + k] = min_idxs[k];
      dists[n * P1 * K + p1 * K + k] = min_dists[k];
    }
  }
}

// This is a shim so we can dispatch using DispatchKernel2D
template <typename scalar_t, int64_t D, int64_t K>
struct KNearestNeighborKernelV2Functor {
  static void run(
      size_t blocks,
      size_t threads,
      const scalar_t* __restrict__ points1,
      const scalar_t* __restrict__ points2,
      const int64_t* __restrict__ lengths1,
      const int64_t* __restrict__ lengths2,
      scalar_t* __restrict__ dists,
      int64_t* __restrict__ idxs,
      const int64_t N,
      const int64_t P1,
      const int64_t P2,
      const size_t norm) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    KNearestNeighborKernelV2<scalar_t, D, K><<<blocks, threads, 0, stream>>>(
        points1, points2, lengths1, lengths2, dists, idxs, N, P1, P2, norm);
  }
};

template <typename scalar_t, int D, int K>
__global__ void KNearestNeighborKernelV3(
    const scalar_t* __restrict__ points1,
    const scalar_t* __restrict__ points2,
    const int64_t* __restrict__ lengths1,
    const int64_t* __restrict__ lengths2,
    scalar_t* __restrict__ dists,
    int64_t* __restrict__ idxs,
    const size_t N,
    const size_t P1,
    const size_t P2,
    const size_t norm) {
  // Same idea as V2, but use register indexing for thread-local arrays.
  // Enabling sorting for this version leads to huge slowdowns; I suspect
  // that it forces min_dists into local memory rather than registers.
  // As a result this version is always unsorted.
  scalar_t cur_point[D];
  scalar_t min_dists[K];
  int min_idxs[K];
  const int64_t chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int64_t chunks_to_do = N * chunks_per_cloud;
  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud;
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    for (int d = 0; d < D; ++d) {
      cur_point[d] = points1[n * P1 * D + p1 * D + d];
    }
    int64_t length2 = lengths2[n];
    RegisterMinK<scalar_t, int, K> mink(min_dists, min_idxs);
    for (int p2 = 0; p2 < length2; ++p2) {
      scalar_t dist = 0;
      for (int d = 0; d < D; ++d) {
        int offset = n * P2 * D + p2 * D + d;
        scalar_t diff = cur_point[d] - points2[offset];
        scalar_t norm_diff = (norm == 2) ? (diff * diff) : ((diff > 0) ? diff : -diff);
        dist += norm_diff;
      }
      mink.add(dist, p2);
    }
    for (int k = 0; k < mink.size(); ++k) {
      idxs[n * P1 * K + p1 * K + k] = min_idxs[k];
      dists[n * P1 * K + p1 * K + k] = min_dists[k];
    }
  }
}

// This is a shim so we can dispatch using DispatchKernel2D
template <typename scalar_t, int64_t D, int64_t K>
struct KNearestNeighborKernelV3Functor {
  static void run(
      size_t blocks,
      size_t threads,
      const scalar_t* __restrict__ points1,
      const scalar_t* __restrict__ points2,
      const int64_t* __restrict__ lengths1,
      const int64_t* __restrict__ lengths2,
      scalar_t* __restrict__ dists,
      int64_t* __restrict__ idxs,
      const size_t N,
      const size_t P1,
      const size_t P2,
      const size_t norm) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    KNearestNeighborKernelV3<scalar_t, D, K><<<blocks, threads, 0, stream>>>(
        points1, points2, lengths1, lengths2, dists, idxs, N, P1, P2, norm);
  }
};

constexpr int V1_MIN_D = 1;
constexpr int V1_MAX_D = 32;

constexpr int V2_MIN_D = 1;
constexpr int V2_MAX_D = 8;
constexpr int V2_MIN_K = 1;
constexpr int V2_MAX_K = 32;

constexpr int V3_MIN_D = 1;
constexpr int V3_MAX_D = 8;
constexpr int V3_MIN_K = 1;
constexpr int V3_MAX_K = 4;

bool InBounds(const int64_t min, const int64_t x, const int64_t max) {
  return min <= x && x <= max;
}

bool KnnCheckVersion(int version, const int64_t D, const int64_t K) {
  if (version == 0) {
    return true;
  } else if (version == 1) {
    return InBounds(V1_MIN_D, D, V1_MAX_D);
  } else if (version == 2) {
    return InBounds(V2_MIN_D, D, V2_MAX_D) && InBounds(V2_MIN_K, K, V2_MAX_K);
  } else if (version == 3) {
    return InBounds(V3_MIN_D, D, V3_MAX_D) && InBounds(V3_MIN_K, K, V3_MAX_K);
  }
  return false;
}

int ChooseVersion(const int64_t D, const int64_t K) {
  for (int version = 3; version >= 1; version--) {
    if (KnnCheckVersion(version, D, K)) {
      return version;
    }
  }
  return 0;
}

std::tuple<at::Tensor, at::Tensor> KNearestNeighborIdxCuda(
    const at::Tensor& p1,
    const at::Tensor& p2,
    const at::Tensor& lengths1,
    const at::Tensor& lengths2,
    const int norm,
    const int K,
    int version) {
  // Check inputs are on the same device
  at::TensorArg p1_t{p1, "p1", 1}, p2_t{p2, "p2", 2},
      lengths1_t{lengths1, "lengths1", 3}, lengths2_t{lengths2, "lengths2", 4};
  at::CheckedFrom c = "KNearestNeighborIdxCuda";
  at::checkAllSameGPU(c, {p1_t, p2_t, lengths1_t, lengths2_t});
  at::checkAllSameType(c, {p1_t, p2_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(p1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const auto N = p1.size(0);
  const auto P1 = p1.size(1);
  const auto P2 = p2.size(1);
  const auto D = p2.size(2);
  const int64_t K_64 = K;

  TORCH_CHECK((norm == 1) || (norm == 2), "Norm must be 1 or 2.");

  TORCH_CHECK(p1.size(2) == D, "Point sets must have the same last dimension");
  auto long_dtype = lengths1.options().dtype(at::kLong);
  auto idxs = at::zeros({N, P1, K}, long_dtype);
  auto dists = at::zeros({N, P1, K}, p1.options());

  if (idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(idxs, dists);
  }

  if (version < 0) {
    version = ChooseVersion(D, K);
  } else if (!KnnCheckVersion(version, D, K)) {
    int new_version = ChooseVersion(D, K);
    std::cout << "WARNING: Requested KNN version " << version
              << " is not compatible with D = " << D << "; K = " << K
              << ". Falling back to version = " << new_version << std::endl;
    version = new_version;
  }

  // At this point we should have a valid version no matter what data the user
  // gave us. But we can check once more to be sure; however this time
  // assert fail since failing at this point means we have a bug in our version
  // selection or checking code.
  AT_ASSERTM(KnnCheckVersion(version, D, K), "Invalid version");

  const size_t threads = 256;
  const size_t blocks = 256;
  if (version == 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        p1.scalar_type(), "knn_kernel_cuda", ([&] {
          KNearestNeighborKernelV0<scalar_t><<<blocks, threads, 0, stream>>>(
              p1.contiguous().data_ptr<scalar_t>(),
              p2.contiguous().data_ptr<scalar_t>(),
              lengths1.contiguous().data_ptr<int64_t>(),
              lengths2.contiguous().data_ptr<int64_t>(),
              dists.data_ptr<scalar_t>(),
              idxs.data_ptr<int64_t>(),
              N,
              P1,
              P2,
              D,
              K,
              norm);
        }));
  } else if (version == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p1.scalar_type(), "knn_kernel_cuda", ([&] {
                                 DispatchKernel1D<
                                     KNearestNeighborV1Functor,
                                     scalar_t,
                                     V1_MIN_D,
                                     V1_MAX_D>(
                                     D,
                                     blocks,
                                     threads,
                                     p1.contiguous().data_ptr<scalar_t>(),
                                     p2.contiguous().data_ptr<scalar_t>(),
                                     lengths1.contiguous().data_ptr<int64_t>(),
                                     lengths2.contiguous().data_ptr<int64_t>(),
                                     dists.data_ptr<scalar_t>(),
                                     idxs.data_ptr<int64_t>(),
                                     N,
                                     P1,
                                     P2,
                                     K,
                                     norm);
                               }));
  } else if (version == 2) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p1.scalar_type(), "knn_kernel_cuda", ([&] {
                                 DispatchKernel2D<
                                     KNearestNeighborKernelV2Functor,
                                     scalar_t,
                                     V2_MIN_D,
                                     V2_MAX_D,
                                     V2_MIN_K,
                                     V2_MAX_K>(
                                     D,
                                     K_64,
                                     blocks,
                                     threads,
                                     p1.contiguous().data_ptr<scalar_t>(),
                                     p2.contiguous().data_ptr<scalar_t>(),
                                     lengths1.contiguous().data_ptr<int64_t>(),
                                     lengths2.contiguous().data_ptr<int64_t>(),
                                     dists.data_ptr<scalar_t>(),
                                     idxs.data_ptr<int64_t>(),
                                     N,
                                     P1,
                                     P2,
                                     norm);
                               }));
  } else if (version == 3) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p1.scalar_type(), "knn_kernel_cuda", ([&] {
                                 DispatchKernel2D<
                                     KNearestNeighborKernelV3Functor,
                                     scalar_t,
                                     V3_MIN_D,
                                     V3_MAX_D,
                                     V3_MIN_K,
                                     V3_MAX_K>(
                                     D,
                                     K_64,
                                     blocks,
                                     threads,
                                     p1.contiguous().data_ptr<scalar_t>(),
                                     p2.contiguous().data_ptr<scalar_t>(),
                                     lengths1.contiguous().data_ptr<int64_t>(),
                                     lengths2.contiguous().data_ptr<int64_t>(),
                                     dists.data_ptr<scalar_t>(),
                                     idxs.data_ptr<int64_t>(),
                                     N,
                                     P1,
                                     P2,
                                     norm);
                               }));
  }
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(idxs, dists);
}

// ------------------------------------------------------------- //
//                   Backward Operators                          //
// ------------------------------------------------------------- //

template <typename scalar_t>
__global__ void KNearestNeighborBackwardKernel(
    const scalar_t* __restrict__ p1, // (N, P1, D)
    const scalar_t* __restrict__ p2, // (N, P2, D)
    const int64_t* __restrict__ lengths1, // (N,)
    const int64_t* __restrict__ lengths2, // (N,)
    const int64_t* __restrict__ idxs, // (N, P1, K)
    const scalar_t* __restrict__ grad_dists, // (N, P1, K)
    scalar_t* __restrict__ grad_p1, // (N, P1, D)
    scalar_t* __restrict__ grad_p2, // (N, P2, D)
    const size_t N,
    const size_t P1,
    const size_t P2,
    const size_t K,
    const size_t D,
    const size_t norm) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  for (size_t i = tid; i < N * P1 * K * D; i += stride) {
    const size_t n = i / (P1 * K * D); // batch index
    size_t rem = i % (P1 * K * D);
    const size_t p1_idx = rem / (K * D); // index of point in p1
    rem = rem % (K * D);
    const size_t k = rem / D; // k-th nearest neighbor
    const size_t d = rem % D; // d-th dimension in the feature vector

    const size_t num1 = lengths1[n]; // number of valid points in p1 in batch
    const size_t num2 = lengths2[n]; // number of valid points in p2 in batch
    if ((p1_idx < num1) && (k < num2)) {
      const scalar_t grad_dist = grad_dists[n * P1 * K + p1_idx * K + k];
      // index of point in p2 corresponding to the k-th nearest neighbor
      const int64_t p2_idx = idxs[n * P1 * K + p1_idx * K + k];
      // If the index is the pad value of -1 then ignore it
      if (p2_idx == -1) {
        continue;
      }
      scalar_t diff = 0.0;
      if (norm == 1) {
        scalar_t sign =
            (p1[n * P1 * D + p1_idx * D + d] > p2[n * P2 * D + p2_idx * D + d])
            ? 1.0
            : -1.0;
        diff = grad_dist * sign;
      } else { // norm is 2
        diff = 2.0 * grad_dist *
            (p1[n * P1 * D + p1_idx * D + d] - p2[n * P2 * D + p2_idx * D + d]);
      }
      gpuAtomicAdd(grad_p1 + n * P1 * D + p1_idx * D + d, diff);
      gpuAtomicAdd(grad_p2 + n * P2 * D + p2_idx * D + d, -1.0f * diff);
    }
  }
}

std::tuple<at::Tensor, at::Tensor> KNearestNeighborBackwardCuda(
    const at::Tensor& p1,
    const at::Tensor& p2,
    const at::Tensor& lengths1,
    const at::Tensor& lengths2,
    const at::Tensor& idxs,
    int norm,
    const at::Tensor& grad_dists) {
  // Check inputs are on the same device
  at::TensorArg p1_t{p1, "p1", 1}, p2_t{p2, "p2", 2},
      lengths1_t{lengths1, "lengths1", 3}, lengths2_t{lengths2, "lengths2", 4},
      idxs_t{idxs, "idxs", 5}, grad_dists_t{grad_dists, "grad_dists", 6};
  at::CheckedFrom c = "KNearestNeighborBackwardCuda";
  at::checkAllSameGPU(
      c, {p1_t, p2_t, lengths1_t, lengths2_t, idxs_t, grad_dists_t});
  at::checkAllSameType(c, {p1_t, p2_t, grad_dists_t});

  // This is nondeterministic because atomicAdd
  at::globalContext().alertNotDeterministic("KNearestNeighborBackwardCuda");

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(p1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const auto N = p1.size(0);
  const auto P1 = p1.size(1);
  const auto P2 = p2.size(1);
  const auto D = p2.size(2);
  const auto K = idxs.size(2);

  TORCH_CHECK(p1.size(2) == D, "Point sets must have the same last dimension");
  TORCH_CHECK(idxs.size(0) == N, "KNN idxs must have the same batch dimension");
  TORCH_CHECK(
      idxs.size(1) == P1, "KNN idxs must have the same point dimension as p1");
  TORCH_CHECK(grad_dists.size(0) == N);
  TORCH_CHECK(grad_dists.size(1) == P1);
  TORCH_CHECK(grad_dists.size(2) == K);

  auto grad_p1 = at::zeros({N, P1, D}, p1.options());
  auto grad_p2 = at::zeros({N, P2, D}, p2.options());

  if (grad_p1.numel() == 0 || grad_p2.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_p1, grad_p2);
  }

  const int blocks = 64;
  const int threads = 512;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(p1.scalar_type(), "knn_backward_kernel_cuda", ([&] {
    KNearestNeighborBackwardKernel<scalar_t><<<blocks, threads, 0, stream>>>(
        p1.contiguous().data_ptr<scalar_t>(),
        p2.contiguous().data_ptr<scalar_t>(),
        lengths1.contiguous().data_ptr<int64_t>(),
        lengths2.contiguous().data_ptr<int64_t>(),
        idxs.contiguous().data_ptr<int64_t>(),
        grad_dists.contiguous().data_ptr<scalar_t>(),
        grad_p1.data_ptr<scalar_t>(),
        grad_p2.data_ptr<scalar_t>(),
        N,
        P1,
        P2,
        K,
        D,
        norm);
  }));

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_p1, grad_p2);
}
