#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <math.h>
#include <cstdio>
#include <sstream>
#include <tuple>
#include "rasterize_points/rasterization_utils.cuh"

namespace {
// A little structure for holding details about a pixel.
struct Pix {
  float z; // Depth of the reference point.
  int32_t idx; // Index of the reference point.
  float dist2; // Euclidean distance square to the reference point.
};

__device__ inline bool operator<(const Pix& a, const Pix& b) {
  return a.z < b.z;
}

// This function checks if a pixel given by xy location pxy lies within the
// point with index p and batch index n. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the points which intersect
// with this pixel sorted by closest z distance. If the pixel pxy lies in the
// point, the list (q) is updated and re-orderered in place. In addition
// the auxiliary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizePointsNaiveCudaKernel and
// RasterizePointsFineCudaKernel.
template <typename PointQ>
__device__ void CheckPixelInsidePoint(
    const float* points, // (P, 3)
    const int p_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    PointQ& q,
    const float* radius,
    const float xf,
    const float yf,
    const int K) {
  const float px = points[p_idx * 3 + 0];
  const float py = points[p_idx * 3 + 1];
  const float pz = points[p_idx * 3 + 2];
  const float p_radius = radius[p_idx];
  const float radius2 = p_radius * p_radius;
  if (pz < 0)
    return; // Don't render points behind the camera
  const float dx = xf - px;
  const float dy = yf - py;
  const float dist2 = dx * dx + dy * dy;
  if (dist2 < radius2) {
    if (q_size < K) {
      // Just insert it
      q[q_size] = {pz, p_idx, dist2};
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = q_size;
      }
      q_size++;
    } else if (pz < q_max_z) {
      // Overwrite the old max, and find the new max
      q[q_max_idx] = {pz, p_idx, dist2};
      q_max_z = pz;
      for (int i = 0; i < K; i++) {
        if (q[i].z > q_max_z) {
          q_max_z = q[i].z;
          q_max_idx = i;
        }
      }
    }
  }
}
} // namespace
// ****************************************************************************
// *                          NAIVE RASTERIZATION                             *
// ****************************************************************************

__global__ void RasterizePointsNaiveCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float* radius,
    const int N,
    const int H,
    const int W,
    const int K,
    int32_t* point_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists) { // (N, H, W, K)
  // Simple version: One thread per output pixel
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = tid; i < N * H * W; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (H * W); // Batch index
    const int pix_idx = i % (H * W);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    // screen coordinates to ndc coordinates of pixel.
    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    // TODO(jcjohns) Abstract this out into a standalone data structure
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t point_start_idx = cloud_to_packed_first_idx[n];
    const int64_t point_stop_idx = point_start_idx + num_points_per_cloud[n];

    for (int p_idx = point_start_idx; p_idx < point_stop_idx; ++p_idx) {
      CheckPixelInsidePoint(
          points, p_idx, q_size, q_max_z, q_max_idx, q, radius, xf, yf, K);
    }
    BubbleSort(q, q_size);
    int idx = n * H * W * K + pix_idx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizePointsNaiveCuda(
    const at::Tensor& points, // (P. 3)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizePointsNaiveCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 3,
      "points must have dimensions (num_points, 3)");
  TORCH_CHECK(
      num_points_per_cloud.size(0) == cloud_to_packed_first_idx.size(0),
      "num_points_per_cloud must have same size first dimension as cloud_to_packed_first_idx");

  const int N = num_points_per_cloud.size(0); // batch size.
  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);
  const int K = points_per_pixel;

  if (K > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  auto int_opts = num_points_per_cloud.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, H, W, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsNaiveCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius.contiguous().data_ptr<float>(),
      N,
      H,
      W,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizePointsFineCudaKernel(
    const float* points, // (P, 3)
    const int32_t* bin_points, // (N, BH, BW, T)
    const float* radius,
    const int bin_size,
    const int N,
    const int BH, // num_bins y
    const int BW, // num_bins x
    const int M,
    const int H,
    const int W,
    const int K,
    int32_t* point_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists) { // (N, H, W, K)
  // This can be more than H * W if H or W are not divisible by bin_size.
  const int num_pixels = N * BH * BW * bin_size * bin_size;
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from points and bin_points.
    int i = pid;
    const int n = i / (BH * BW * bin_size * bin_size);
    i %= BH * BW * bin_size * bin_size;
    const int by = i / (BW * bin_size * bin_size);
    i %= BW * bin_size * bin_size;
    const int bx = i / (bin_size * bin_size);
    i %= bin_size * bin_size;

    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= H || xi >= W)
      continue;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    // This part looks like the naive rasterization kernel, except we use
    // bin_points to only look at a subset of points already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; ++m) {
      const int p = bin_points[n * BH * BW * M + by * BW * M + bx * M + m];
      if (p < 0) {
        // bin_points uses -1 as a sentinal value
        continue;
      }
      CheckPixelInsidePoint(
          points, p, q_size, q_max_z, q_max_idx, q, radius, xf, yf, K);
    }
    // Now we've looked at all the points for this bin, so we can write
    // output for the current pixel.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const int pix_idx = n * H * W * K + yidx * W * K + xidx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizePointsFineCuda(
    const at::Tensor& points, // (P, 3)
    const at::Tensor& bin_points,
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int bin_size,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      bin_points_t{bin_points, "bin_points", 2};
  at::CheckedFrom c = "RasterizePointsFineCuda";
  at::checkAllSameGPU(c, {points_t, bin_points_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = bin_points.size(0);
  const int BH = bin_points.size(1);
  const int BW = bin_points.size(2);
  const int M = bin_points.size(3);
  const int K = points_per_pixel;

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 150");
  }
  auto int_opts = bin_points.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, H, W, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsFineCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      bin_points.contiguous().data_ptr<int32_t>(),
      radius.contiguous().data_ptr<float>(),
      bin_size,
      N,
      BH,
      BW,
      M,
      H,
      W,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO(T55115174) Add more documentation for backward kernel.
__global__ void RasterizePointsBackwardCudaKernel(
    const float* points, // (P, 3)
    const int32_t* idxs, // (N, H, W, K)
    const int N,
    const int P,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_dists, // (N, H, W, K)
    float* grad_points) { // (P, 3)
  // Parallelized over each of K points per pixel, for each pixel in images of
  // size H * W, for each image in the batch of size N.
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < N * H * W * K; i += num_threads) {
    // const int n = i / (H * W * K); // batch index (not needed).
    const int yxk = i % (H * W * K);
    const int yi = yxk / (W * K);
    const int xk = yxk % (W * K);
    const int xi = xk / K;
    // k = xk % K (We don't actually need k, but this would be it.)
    // Reverse ordering of X and Y axes.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const float xf = PixToNonSquareNdc(xidx, W, H);
    const float yf = PixToNonSquareNdc(yidx, H, W);

    const int p = idxs[i];
    if (p < 0)
      continue;
    const float grad_dist2 = grad_dists[i];
    const int p_ind = p * 3; // index into packed points tensor
    const float px = points[p_ind + 0];
    const float py = points[p_ind + 1];
    const float dx = px - xf;
    const float dy = py - yf;
    const float grad_px = 2.0f * grad_dist2 * dx;
    const float grad_py = 2.0f * grad_dist2 * dy;
    const float grad_pz = grad_zbuf[i];
    atomicAdd(grad_points + p_ind + 0, grad_px);
    atomicAdd(grad_points + p_ind + 1, grad_py);
    atomicAdd(grad_points + p_ind + 2, grad_pz);
  }
}

at::Tensor RasterizePointsBackwardCuda(
    const at::Tensor& points, // (N, P, 3)
    const at::Tensor& idxs, // (N, H, W, K)
    const at::Tensor& grad_zbuf, // (N, H, W, K)
    const at::Tensor& grad_dists) { // (N, H, W, K)

  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1}, idxs_t{idxs, "idxs", 2},
      grad_zbuf_t{grad_zbuf, "grad_zbuf", 3},
      grad_dists_t{grad_dists, "grad_dists", 4};
  at::CheckedFrom c = "RasterizePointsBackwardCuda";
  at::checkAllSameGPU(c, {points_t, idxs_t, grad_zbuf_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, grad_zbuf_t, grad_dists_t});
  // This is nondeterministic because atomicAdd
  at::globalContext().alertNotDeterministic("RasterizePointsBackwardCuda");
  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int P = points.size(0);
  const int N = idxs.size(0);
  const int H = idxs.size(1);
  const int W = idxs.size(2);
  const int K = idxs.size(3);

  at::Tensor grad_points = at::zeros({P, 3}, points.options());

  if (grad_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_points;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizePointsBackwardCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      idxs.contiguous().data_ptr<int32_t>(),
      N,
      P,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_points.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return grad_points;
}
