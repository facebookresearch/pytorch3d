#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void weightedSumCudaForwardKernel(
    // clang-format off
    at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> result,
    const at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> features,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> alphas,
    const at::PackedTensorAccessor64<int64_t, 4, at::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * H * W;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Parallelize over each feature in each pixel in images of size H * W,
  // for each image in the batch of size batch_size
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (H * W);
    int j = (pid % (H * W)) / W;
    int i = (pid % (H * W)) % W;

    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];
      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }

      // Accumulate the values
      float alpha = alphas[batch][k][j][i];
      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(&result[batch][ch][j][i], features[ch][n_idx] * alpha);
    }
  }
}

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void weightedSumCudaBackwardKernel(
    // clang-format off
    at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> grad_features,
    at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> grad_alphas,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> grad_outputs,
    const at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> features,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> alphas,
    const at::PackedTensorAccessor64<int64_t, 4, at::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * H * W;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Iterate over each pixel to compute the contribution to the
  // gradient for the features and weights
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (H * W);
    int j = (pid % (H * W)) / W;
    int i = (pid % (H * W)) % W;

    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];
      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }
      float alpha = alphas[batch][k][j][i];

      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(
          &grad_alphas[batch][k][j][i],
          features[ch][n_idx] * grad_outputs[batch][ch][j][i]);
      atomicAdd(
          &grad_features[ch][n_idx], alpha * grad_outputs[batch][ch][j][i]);
    }
  }
}

at::Tensor weightedSumCudaForward(
    const at::Tensor& features,
    const at::Tensor& alphas,
    const at::Tensor& points_idx) {
  // Check inputs are on the same device
  at::TensorArg features_t{features, "features", 1},
      alphas_t{alphas, "alphas", 2}, points_idx_t{points_idx, "points_idx", 3};
  at::CheckedFrom c = "weightedSumCudaForward";
  at::checkAllSameGPU(c, {features_t, alphas_t, points_idx_t});
  at::checkAllSameType(c, {features_t, alphas_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t batch_size = points_idx.size(0);
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  auto result = at::zeros({batch_size, C, H, W}, features.options());

  if (result.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return result;
  }

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(batch_size, 1024 / batch_size + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  weightedSumCudaForwardKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
      // clang-format off
      // As we are using packed accessors here the tensors
      // do not need to be made contiguous.
      result.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, at::RestrictPtrTraits>());
  // clang-format on
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

std::tuple<at::Tensor, at::Tensor> weightedSumCudaBackward(
    const at::Tensor& grad_outputs,
    const at::Tensor& features,
    const at::Tensor& alphas,
    const at::Tensor& points_idx) {
  // Check inputs are on the same device
  at::TensorArg grad_outputs_t{grad_outputs, "grad_outputs", 1},
      features_t{features, "features", 2}, alphas_t{alphas, "alphas", 3},
      points_idx_t{points_idx, "points_idx", 4};
  at::CheckedFrom c = "weightedSumCudaBackward";
  at::checkAllSameGPU(c, {grad_outputs_t, features_t, alphas_t, points_idx_t});
  at::checkAllSameType(c, {grad_outputs_t, features_t, alphas_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto grad_features = at::zeros_like(features);
  auto grad_alphas = at::zeros_like(alphas);

  if (grad_features.numel() == 0 || grad_alphas.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_features, grad_alphas);
  }

  const int64_t bs = points_idx.size(0);

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(bs, 1024 / bs + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  weightedSumCudaBackwardKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
      // clang-format off
      // As we are using packed accessors here the tensors
      // do not need to be made contiguous.
      grad_features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      grad_alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      grad_outputs.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, at::RestrictPtrTraits>());
  // clang-format on
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_features, grad_alphas);
}
