#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

__constant__ const float kEpsilon = 1e-9;

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void alphaCompositeCudaForwardKernel(
    // clang-format off
    at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> result,
    const at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> features,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> alphas,
    const at::PackedTensorAccessor64<int64_t, 4, at::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * H * W;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Iterate over each feature in each pixel
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (H * W);
    int j = (pid % (H * W)) / W;
    int i = (pid % (H * W)) % W;

    // alphacomposite the different values
    float cum_alpha = 1.;
    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];

      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }

      float alpha = alphas[batch][k][j][i];
      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(
          &result[batch][ch][j][i], features[ch][n_idx] * cum_alpha * alpha);
      cum_alpha = cum_alpha * (1 - alpha);
    }
  }
}

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void alphaCompositeCudaBackwardKernel(
    // clang-format off
    at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> grad_features,
    at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> grad_alphas,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> grad_outputs,
    const at::PackedTensorAccessor64<float, 2, at::RestrictPtrTraits> features,
    const at::PackedTensorAccessor64<float, 4, at::RestrictPtrTraits> alphas,
    const at::PackedTensorAccessor64<int64_t, 4, at::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * H * W;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Parallelize over each feature in each pixel in images of size H * W,
  // for each image in the batch of size batch_size
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (H * W);
    int j = (pid % (H * W)) / W;
    int i = (pid % (H * W)) % W;

    // alphacomposite the different values
    float cum_alpha = 1.;
    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];

      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }
      float alpha = alphas[batch][k][j][i];

      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(
          &grad_alphas[batch][k][j][i],
          cum_alpha * features[ch][n_idx] * grad_outputs[batch][ch][j][i]);
      atomicAdd(
          &grad_features[ch][n_idx],
          cum_alpha * alpha * grad_outputs[batch][ch][j][i]);

      // Iterate over all (K-1) nearest points to update gradient
      for (int t = 0; t < k; ++t) {
        int t_idx = points_idx[batch][t][j][i];
        // Sentinel value is -1, indicating no point overlaps this pixel
        if (t_idx < 0) {
          continue;
        }
        float alpha_tvalue = alphas[batch][t][j][i];
        // TODO(gkioxari) It might be more efficient to have threads write in a
        // local variable, and move atomicAdd outside of the loop such that
        // atomicAdd is executed once per thread.
        atomicAdd(
            &grad_alphas[batch][t][j][i],
            -grad_outputs[batch][ch][j][i] * features[ch][n_idx] * cum_alpha *
                alpha / (1 - alpha_tvalue + kEpsilon));
      }

      cum_alpha = cum_alpha * (1 - alphas[batch][k][j][i]);
    }
  }
}

at::Tensor alphaCompositeCudaForward(
    const at::Tensor& features,
    const at::Tensor& alphas,
    const at::Tensor& points_idx) {
  // Check inputs are on the same device
  at::TensorArg features_t{features, "features", 1},
      alphas_t{alphas, "alphas", 2}, points_idx_t{points_idx, "points_idx", 3};
  at::CheckedFrom c = "alphaCompositeCudaForward";
  at::checkAllSameGPU(c, {features_t, alphas_t, points_idx_t});
  at::checkAllSameType(c, {features_t, alphas_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t batch_size = points_idx.size(0);
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  auto result = at::zeros({batch_size, C, H, W}, features.options());

  if (result.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return result;
  }

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(batch_size, 1024 / batch_size + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  alphaCompositeCudaForwardKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
      // clang-format off
      // As we are using packed accessors here the tensors
      // do not need to be made contiguous.
      result.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, at::RestrictPtrTraits>());
  // clang-format on
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

std::tuple<at::Tensor, at::Tensor> alphaCompositeCudaBackward(
    const at::Tensor& grad_outputs,
    const at::Tensor& features,
    const at::Tensor& alphas,
    const at::Tensor& points_idx) {
  // Check inputs are on the same device
  at::TensorArg grad_outputs_t{grad_outputs, "grad_outputs", 1},
      features_t{features, "features", 2}, alphas_t{alphas, "alphas", 3},
      points_idx_t{points_idx, "points_idx", 4};
  at::CheckedFrom c = "alphaCompositeCudaBackward";
  at::checkAllSameGPU(c, {grad_outputs_t, features_t, alphas_t, points_idx_t});
  at::checkAllSameType(c, {grad_outputs_t, features_t, alphas_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto grad_features = at::zeros_like(features);
  auto grad_alphas = at::zeros_like(alphas);

  if (grad_features.numel() == 0 || grad_alphas.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_features, grad_alphas);
  }

  const int64_t bs = alphas.size(0);

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(bs, 1024 / bs + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  alphaCompositeCudaBackwardKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
      // clang-format off
      // As we are using packed accessors here the tensors
      // do not need to be made contiguous.
      grad_features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      grad_alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      grad_outputs.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, at::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, at::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, at::RestrictPtrTraits>());
  // clang-format on
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_features, grad_alphas);
}
